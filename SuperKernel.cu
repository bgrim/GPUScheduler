#include "hip/hip_runtime.h"
#include <stdio.h>

//#include "Queues/QueueJobs.cu"
#include "Kernels/Sleep0.cu"
#include "Kernels/Sleep1.cu"
#include "Kernels/AddSleep.cu"
#include "Kernels/MatrixSquare.cu"


__device__ JobDescription *executeJob(JobDescription *currentJob);

__global__ void superKernel(Queue incoming, Queue results, int numJobsPerWarp)
{ 
    // init and result are arrays of integers where result should end up
    // being the result of incrementing all elements of init.
    // They have n elements and are (n+1) long. The should wait for the
    // first element to be set to zero
    int warp_size = 32;

    int threadID = threadIdx.x % warp_size;
    int warpID = threadIdx.x / warp_size;   //added depenency on block

    __shared__ JobDescription currentJobs[32];

    //    int numJobsPerWarp = 1;
    int i;
    for(i=0;i<numJobsPerWarp;i++)
    {
      if(threadID==0) FrontAndDequeueJob(incoming, &currentJobs[warpID]);

      JobDescription *retval;
      if(threadID<(currentJobs[warpID].numThreads)) retval = executeJob(&currentJobs[warpID]);

      if(threadID==0) EnqueueResult(retval, results);
    }
}

__device__ JobDescription *executeJob(JobDescription *currentJob){

  int JobType = currentJob->JobType;

  //  int SleepTime = 5000;
  int clockRate = 1560000;

  // large switch
  switch(JobType){
    case 0:
      sleep0(currentJob->params, clockRate);
      break;
    case 1:
      sleep1(currentJob->params, clockRate);
      break;
    case 2:
      addSleep(currentJob->params);
      break;
    case 3:
      matrixSquare(currentJob->params);
      break;
  }
  return currentJob;
}

