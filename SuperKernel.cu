#include "hip/hip_runtime.h"
#include <stdio.h>

//#include "Queues/QueueJobs.cu"
#include "Kernels/Sleep0.cu"
#include "Kernels/Sleep1.cu"
#include "Kernels/AddSleep.cu"

__device__ JobDescription executeJob(JobDescription currentJob);

__global__ void superKernel(Queue incoming, Queue results)
{ 
    // init and result are arrays of integers where result should end up
    // being the result of incrementing all elements of init.
    // They have n elements and are (n+1) long. The should wait for the
    // first element to be set to zero
    int warp_size = 32;

    int threadID = threadIdx.x % warp_size;
    //int warpID = threadIdx.x / warp_size;   //added depenency on block

    int numJobs = 1;
    int i;

    //int clockRate = 1560000;
    //int sleep = 1000;
    //sleep0(&sleep, clockRate);

    for(i=0;i<numJobs;i++)
    {
      JobDescription currentJob;

      if(threadID==0) currentJob = FrontAndDequeueJob(incoming);

      JobDescription retval;
      if(threadID<(currentJob.numThreads)) retval = executeJob(currentJob);

      if(threadID==0) EnqueueResult(retval, results);
    }
}

__device__ JobDescription executeJob(JobDescription currentJob){

  int JobType = currentJob.JobType;

  //int SleepTime = 1000;
  int clockRate = 1560000;

  // large switch
  switch(JobType){
    case 0:
      sleep0(currentJob.params, clockRate);
      break;
    case 1:
      sleep1(currentJob.params, clockRate);
      break;
    case 2:
      addSleep(currentJob.params);
      break;
  }
  return currentJob;
}

