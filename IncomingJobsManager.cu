#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#include "Queues/QueueJobs.cu"
#include "Queues/QueueResults.cu"


pthread_t start_IncomingJobsManager(Queue d_newJobs)
{
//This should do any initializing that the incoming jobs
//  manager will need and then launch a thread running
//  main_IncomingJobsManager(  ), returning that thread

  pthread_t thread1;
  pthread_create( &thread1, NULL, main_IncomingJobsManager, (void*) d_newJobs);
}


void *main_IncomingJobsManager(void *p)
{
//The thread should get job descriptions some how and Enqueue them
//  into the queue in params
//    --eventually this should get jobs from an external application
//      but will probably just be hardcoded at first or a parameter

  Queue d_newJobs = (Queue) p;

  // Hard code for testing
  int HC_JobType = 0; // hard code the job type for sleeps
  int HC_JobID;
  void* HC_params;
  int HC_numThreads = 32;
  int HC_jobs = 64;

  int size = sizeof(struct JobDescription);

  int i;
  for(i=0;i<HC_jobs;i++){
    HC_JobID = i;
    // launch queue jobs
    // malloc the host structure
    JobDescription h_JobDescription = malloc(size);

    // set the values to the host structure
    h_JobDescription->JobType = HC_JobType;
    h_JobDescription->JobID = HC_JobID;
    h_JobDescription->params = HC_params;
    h_JobDescription->numThreads = HC_numThreads;

    JobDescription d_JobDescription;



    // cuda Malloc for the structure
    hipMalloc(&d_JobDescription, size);

    // cuda mem copy
    hipMemcpy(&d_JobDescription, &h_JobDescription, size, hipMemcpyHostToDevice); // maybe we can test this later with async

    // enqueue jobs
    EnqueueJob(d_JobDescription, d_newJobs);

    // free the local memory
    free(h_JobDescription);
  }
}
