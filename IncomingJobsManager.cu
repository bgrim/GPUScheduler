#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

//#include "Queues/QueueJobs.cu"
void *main_IncomingJobsManager(void *p);

pthread_t start_IncomingJobsManager(Queue d_newJobs)
{
//This should do any initializing that the incoming jobs
//  manager will need and then launch a thread running
//  main_IncomingJobsManager(  ), returning that thread

  pthread_t thread1;
  pthread_create( &thread1, NULL, main_IncomingJobsManager, (void*) d_newJobs);
  return thread1;
}


void *main_IncomingJobsManager(void *p)
{
//The thread should get job descriptions some how and Enqueue them
//  into the queue in params
//    --eventually this should get jobs from an external application
//      but will probably just be hardcoded at first or a parameter

  Queue d_newJobs = (Queue) p;

  // Hard code for testing
  int HC_JobType = 0; // hard code the job type for sleeps
  int HC_JobID;
  void* HC_params;
  int HC_numThreads = 1;
  int HC_jobs = 1;

  int size = sizeof(struct JobDescription);

  int i;
  for(i=0;i<HC_jobs;i++){
    HC_JobID = i;
    // launch queue jobs
    // malloc the host structure
    JobDescription *h_JobDescription = (JobDescription *) malloc(size);

    // set the values to the host structure
    h_JobDescription->JobType = HC_JobType;
    h_JobDescription->JobID = HC_JobID;
    h_JobDescription->params = HC_params;
    h_JobDescription->numThreads = HC_numThreads;

    JobDescription *d_JobDescription;

    // cuda Malloc for the structure
    hipError_t e1 = hipMalloc((void **) &d_JobDescription, size);
    printf("CUDA ERROR in Enqueue hipMalloc: %s\n", hipGetErrorString(e1));

    // cuda mem copy
    hipError_t e2 = hipMemcpy(d_JobDescription, h_JobDescription, size, hipMemcpyHostToDevice); // maybe we can test this later with async
    printf("CUDA ERROR in Enqueue in hipMemcpy: %s\n", hipGetErrorString(e2));

    printf("Starting Enqueuing job # %d\n", HC_JobID);
    // enqueue jobs
    EnqueueJob(d_JobDescription, d_newJobs);

    printf("Finished Enqueuing job # %d\n", HC_JobID);

    // free the local memory
    free(h_JobDescription);
  }
  return 0;
}
