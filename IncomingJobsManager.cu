#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <sys/time.h>

struct timeval tp;


double getTime_usec() {
    gettimeofday(&tp, NULL);
    return static_cast<double>(tp.tv_sec) * 1E6
            + static_cast<double>(tp.tv_usec);
}


//#include "Queues/QueueJobs.cu"
void *main_IncomingJobsManager(void *p);

pthread_t start_IncomingJobsManager(Queue d_newJobs)
{
//This should do any initializing that the incoming jobs
//  manager will need and then launch a thread running
//  main_IncomingJobsManager(  ), returning that thread

  pthread_attr_t attr;
  pthread_attr_init(&attr);
  pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_JOINABLE);

  pthread_t thread1;
  pthread_create( &thread1, &attr, main_IncomingJobsManager, (void*) d_newJobs);
  pthread_attr_destroy(&attr);
  return thread1;
}


void *moveToCuda(void *val, int size){
  void *ret;
  hipError_t e = hipMalloc(&ret, size);
  if(e!=hipSuccess)printf("CUDA Malloc Error: %s  in  moveToCuda\n", hipGetErrorString (e));
  cudaSafeMemcpy(ret, val, size, 
                 hipMemcpyHostToDevice, stream_dataIn, 
                 "in moveToCuda of IncomingJobsManager.cu");
  return ret;
}

float *makeMatrix(){
  int ROW = 32;
  int COLUMN = 32;

  int a=0, b=0;

  float *stuff = (float *) malloc(2*(COLUMN * ROW * sizeof(float)));
  for(a=0; a<ROW;a++)
    {
      for(b=0; b<COLUMN;b++)
        {
	  stuff[a + b * ROW]=((float)rand())/((float) RAND_MAX);
	  stuff[a + b * ROW + ROW * COLUMN] = 0.0;
	}
    }
  return stuff;
}


void *main_IncomingJobsManager(void *p)
{
//The thread should get job descriptions some how and Enqueue them
//  into the queue in params
//    --eventually this should get jobs from an external application
//      but will probably just be hardcoded at first or a parameter

  Queue d_newJobs = (Queue) p;

  // Hard code for testing
  int HC_JobType = 2; // hard code the job type for sleeps
  int HC_JobID;
  int HC_numThreads = 32;
  int HC_jobs = NUMBER_OF_JOBS;
  //  int HC_matrixWidth = 32;
  //int HC_matrixSize = HC_matrixWidth * HC_matrixWidth;

  int size = sizeof(struct JobDescription);

  printf("Starting IncomingJobs Manager\n");

  void * d_sleep_time = moveToCuda(&SLEEP_TIME, sizeof(int));

  int i;
  for(i=0;i<HC_jobs;i++){
    HC_JobID = i;
    // launch queue jobs
    // malloc the host structure
    JobDescription *h_JobDescription = (JobDescription *) malloc(size);

    // set the values to the host structure
    h_JobDescription->JobType = HC_JobType;
    h_JobDescription->JobID = HC_JobID;

    h_JobDescription->params = d_sleep_time; //AddSleep
    //h_JobDescription->params = moveToCuda(makeMatrix(), (2 * sizeof(float) * HC_matrixSize)); //Matrix
    h_JobDescription->numThreads = HC_numThreads;

    // enqueue jobs
    EnqueueJob(h_JobDescription, d_newJobs);
    //printf("Finished EnqueueJob # %d\n", HC_JobID);

    // free the local memory
    free(h_JobDescription);
  }
  printf("Finished Incoming Jobs Manager\n");
  return 0;
}





