#include "hip/hip_runtime.h"
#include <stdlib.h>

struct JobDescription{
  int JobID;
  int JobType;
  int numThreads;
  void* params;
};

struct QueueRecord {
  JobDescription *Array; //Order matters here, we should improve this later
  int Capacity;          // by having two different Queues with different Orders
  int Rear;
  int Front;
  int ReadLock;
};

typedef QueueRecord *Queue;


////////////////////////////////////////////////////////////
// Locking Functions used to Sync warps access to Queues
////////////////////////////////////////////////////////////
__device__ void getLock(volatile Queue Q)
{
  while(atomicCAS(&(Q->ReadLock), 0, 1) != 0);
}

__device__ void releaseLock(volatile Queue Q)
{
  atomicExch(&(Q->ReadLock),0);
}

///////////////////////////////////////////////////////////
// Device Helper Functions
///////////////////////////////////////////////////////////

__device__ int d_IsEmpty(volatile Queue Q) {
  volatile int *s = &(Q->Rear);
  return (*s+1)%Q->Capacity == Q->Front;
}

__device__ int d_IsFull(volatile Queue Q) {
  volatile int *s = &(Q->Rear);
  return (*s+2)%Q->Capacity == Q->Front;
}


//////////////////////////////////////////////////////////
// Host Helper Functions
//////////////////////////////////////////////////////////
int h_IsEmpty(Queue Q) {
  return (Q->Rear+1)%Q->Capacity == Q->Front;
}

int h_IsFull(Queue Q) {
  return (Q->Rear+2)%Q->Capacity == Q->Front;
}

void *movePointer(void *p, int n){
   char * ret = (char *) p;
   return ((void *)(ret+n));
}

void printAnyErrors()
{
  hipError_t e = hipGetLastError();
  if(e!=hipSuccess){
    printf("CUDA Error: %s\n", hipGetErrorString( e ) );
  }
}

void synchronizeAndPrint(hipStream_t stream, char *s){
  hipError_t e = hipStreamSynchronize(stream);
  if(e!=hipSuccess){
    printf("CUDA Error:   %s   at %s\n", hipGetErrorString( e ), s);
  }
}

////////////////////////////////////////////////////////////
// Constructor and Deconsturctor
////////////////////////////////////////////////////////////

Queue CreateQueue(int MaxElements) {
  Queue Q;

  //if (MaxElements < MinQueueSize) {
    //FatalError("CreateQueueJobs Error: Queue size is too small.");
  //}

  Q = (Queue) malloc (sizeof(struct QueueRecord));
  //if (Q == NULL) {
  //FatalError("CreateQueueJobs Error: Unable to allocate more memory.");
  //}

  hipMalloc((void **)&(Q->Array), sizeof(JobDescription)*MaxElements);

  Q->Capacity = MaxElements;
  Q->Front = 1;
  Q->Rear = 0;
  Q->ReadLock = 0;

  Queue d_Q;
  hipMalloc(&d_Q, sizeof(struct QueueRecord));
  hipMemcpy(d_Q, Q, sizeof(struct QueueRecord), hipMemcpyHostToDevice);
  free(Q);

/*
  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));
  hipMemcpy(h_Q, Q, sizeof(struct QueueRecord), hipMemcpyDeviceToHost);

  printf("  Capacity, %d\n", h_Q->Capacity);
  printf("  Rear,     %d\n", h_Q->Rear);
  printf("  Front,    %d\n", h_Q->Front);
*/

  return d_Q;
}

void DisposeQueue(Queue Q) {
  hipFree(Q);
}

////////////////////////////////////////////////////////////
// Functions to modify a new jobs queue
////////////////////////////////////////////////////////////

void EnqueueJob(JobDescription *h_JobDescription, Queue Q) {
//called by CPU

  int copySize= sizeof(struct QueueRecord);

  //printf("Start of EnqueueJob\n");

  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));
  hipMemcpyAsync(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataIn);
  synchronizeAndPrint(stream_dataIn, "EnqueueJob, Getting Queue");

  printf("Queue Values at Enqueue\n");
  printf("  Capacity, %d\n", h_Q->Capacity);
  printf("  Rear,     %d\n", h_Q->Rear);
  printf("  Front,    %d\n\n", h_Q->Front);

  while(h_IsFull(h_Q)){
    hipMemcpyAsync(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataIn);
    synchronizeAndPrint(stream_dataIn, "EnqueueJob, Getting Queue again...");
  }

  h_Q->Rear = (h_Q->Rear+1)%(h_Q->Capacity);

  //printf("Middle of EnqueueJob\n");

  hipMemcpyAsync(h_Q->Array + h_Q->Rear,
                  h_JobDescription, 
                  sizeof(JobDescription),
                  hipMemcpyHostToDevice, 
                  stream_dataIn);
  synchronizeAndPrint(stream_dataIn, "EnqueueJob, Writing Job Description");


  //printf("End of EnqueueJob\n");

  hipMemcpyAsync(movePointer(Q, 12), movePointer(h_Q, 12), 
                   sizeof(int), hipMemcpyHostToDevice, stream_dataIn);
    synchronizeAndPrint(stream_dataIn, "EnqueueJob, Updating Queue");

  free(h_Q);
/*
  hipMemcpyAsync(h_JobDescription,
		  h_Q->Array + (h_Q->Rear)*sizeof(JobDescription),
                  sizeof(JobDescription),
                  hipMemcpyDeviceToHost,
                  stream_dataIn);
  printf("num of threads = %d\n", h_JobDescription->numThreads);
*/
}

__device__ JobDescription FrontJob(volatile Queue Q) {
//called by GPU
  getLock(Q);

  int count = 0;
  while(d_IsEmpty(Q))count++; //wait for a job

  JobDescription result = Q->Array[Q->Front];
  releaseLock(Q);
  return result;

}

__device__ void DequeueJob(volatile Queue Q) {
//called by GPU
  getLock(Q);

  int count =0;
  while(d_IsEmpty(Q))count++; //wait for a job

  Q->Front = (Q->Front+1)%(Q->Capacity);

  releaseLock(Q);
}

__device__ JobDescription FrontAndDequeueJob(volatile Queue Q) {
//called by GPU
  getLock(Q);

  //d_WaitIsEmpty(Q); //wait for a job

  int count = 0;
  while(d_IsEmpty(Q))count++;

  JobDescription result = Q->Array[Q->Front];
  Q->Front = (Q->Front+1)%(Q->Capacity);

  releaseLock(Q);

  return result;
}

__device__ void EnqueueResult(JobDescription X, volatile Queue Q) {
//called by GPU
  getLock(Q);

  int count =0;
  while(d_IsFull(Q))count++;

  Q->Rear = (Q->Rear+1)%(Q->Capacity);
  Q->Array[Q->Rear] = X;

  releaseLock(Q);
}

JobDescription FrontResult(Queue Q) {
//called by CPU
  int copySize= sizeof(struct QueueRecord);

  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));
  hipMemcpyAsync(h_Q, Q, copySize, hipMemcpyDeviceToHost,stream_dataOut);
  hipStreamSynchronize(stream_dataOut);

  while(h_IsEmpty(h_Q)){
    hipMemcpyAsync(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut);
    hipStreamSynchronize(stream_dataOut);
  }

  return h_Q->Array[h_Q->Front];
}

void DequeueResult(Queue Q) {
//called by CPU
  int copySize= sizeof(struct QueueRecord); 

  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));
  hipMemcpyAsync(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut);
  hipStreamSynchronize(stream_dataOut);

  while(h_IsEmpty(h_Q)){
    hipMemcpyAsync(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut);
    hipStreamSynchronize(stream_dataOut);
  }

  h_Q->Front = (h_Q->Front+1)%(h_Q->Capacity);

  hipMemcpyAsync(movePointer(Q, 16), movePointer(h_Q, 16), 
                   sizeof(int), hipMemcpyHostToDevice, stream_dataOut);
  hipStreamSynchronize(stream_dataOut);
}

JobDescription FrontAndDequeueResult(Queue Q) {
//called by CPU
  //printf("starting fandDresults\n");
  int copySize= sizeof(struct QueueRecord);
  //printf("malloc queue\n");
  Queue h_Q = (Queue) malloc(sizeof(struct QueueRecord));
  //printf("mem cpy\n");
  hipMemcpyAsync(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut);
  synchronizeAndPrint(stream_dataOut, "FandDJob, Getting Queue");

  // printf("%d\n", h_Q->Size);
/*
  printf("Queue Values at Dequeue\n");
  printf("  Capacity, %d\n", h_Q->Capacity);
  printf("  Rear,     %d\n", h_Q->Rear);
  printf("  Front,    %d\n", h_Q->Front);
*/
  while(h_IsEmpty(h_Q)){
    hipMemcpyAsync(h_Q, Q, copySize, hipMemcpyDeviceToHost, stream_dataOut);
              //printf("%d\n", h_Q->Size);
      synchronizeAndPrint(stream_dataOut, "FandDJob, Getting Queue again...");
/*
    printf("Queue Values at Dequeue\n");
    printf("  Capacity, %d\n", h_Q->Capacity);
    printf("  Rear,     %d\n", h_Q->Rear);
    printf("  Front,    %d\n", h_Q->Front);
*/
  }

  //printf("broke out of loop\n");
  JobDescription *result = (JobDescription *) malloc(sizeof(JobDescription));
  //  hipMemcpyAsync(result, h_Q->Array + (h_Q->Front)*sizeof(JobDescription), sizeof(JobDescription), hipMemcpyDeviceToHost, stream_dataOut);
  hipMemcpyAsync(result, &h_Q->Array[h_Q->Front], sizeof(JobDescription), hipMemcpyDeviceToHost, stream_dataOut);
  synchronizeAndPrint(stream_dataOut, "FandDJob, Getting Job Description");

  h_Q->Front = (h_Q->Front+1)%(h_Q->Capacity);
  //printf("Update Device Queue\n");
  hipMemcpyAsync(movePointer(Q, 16), movePointer(h_Q, 16), 
                   sizeof(int), hipMemcpyHostToDevice, stream_dataOut);
  synchronizeAndPrint(stream_dataOut, "FandDJob, Updating Queue");

  return *result;
}
