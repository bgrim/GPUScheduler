#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
//#include "Queues/QueueJobs.cu"

void *main_ResultsManager(void *params);

pthread_t start_ResultsManager(Queue CompletedJobDescriptions)
{
//This should do any initializing that the results manager will
//  need and then launch a thread running main_ResultsManager,
//  returning this thread

  pthread_attr_t attr;
  pthread_attr_init(&attr);
  pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_JOINABLE);

  pthread_t thread2;
  pthread_create( &thread2, NULL, main_ResultsManager, (void*) CompletedJobDescriptions);

  return thread2;
}


void *main_ResultsManager(void *params)
{
//The thread should read results from the queue in param and
//  print something about them to the screen.
//    --eventually this should return the result to the application
//      that requested the work.
  printf("Starting ResultsManager\n"); 

  int HC_jobs = NUMBER_OF_JOBS;
  int i;
  JobDescription currentJob;
  Queue results = (Queue)params;
  
  for(i=0;i<HC_jobs;i++){
    // front and dequeue results
//    printf("Starting to dequeue\n");
    currentJob = FrontAndDequeueResult(results);
/*
    printf("\nJob Finsihed:\n");
    printf("  ID # %d\n", currentJob.JobID);
    printf("  type %d\n", currentJob.JobType);
    printf("  numT %d\n\n", currentJob.numThreads);
*/
    hipFree(&currentJob);
  }
  return 0;
}
