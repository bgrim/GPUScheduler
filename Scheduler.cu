#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

hipStream_t stream_dataIn, stream_dataOut; //this is probably not the best way to make these
                                            // streams globally known, oh well.

#include "Queues/QueueJobs.cu"
#include "IncomingJobsManager.cu"
#include "ResultsManager.cu"
#include "SuperKernel.cu"

////////////////////////////////////////////////////////////////////
// The Main
////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{
//Define constants
  int warp_size = 32;

  int warps = 1;   //possible input arguements
  int blocks = 1;
  
  dim3 threads(warp_size*warps/blocks, 1);
  dim3 grid(blocks, 1);

//Allocate streams, Queues
  /* Notes on these structures:
       Streams: one stream for each direction of data movement and one
                    for the Super Kernel.
       QueueJobs: The Scheduler will Enqueue jobDescriptions
                  The Super Kernel will Dequeue and execute them
       QueueResults: The Super Kernel will Enqueue jobResults
                     The Scheduler will Dequeue them and send results to its caller
  */
  hipStream_t stream_kernel;
  hipStreamCreate(&stream_kernel);
  hipStreamCreate(&stream_dataIn);
  hipStreamCreate(&stream_dataOut);

  Queue d_newJobs = CreateQueue(128); //FIX, make this use stream_dataIn

  Queue d_finishedJobs = CreateQueue(128); //FIX, make this use stream_dataOut


//Launch the super kernel
  superKernel<<< grid, threads, 0, stream_kernel>>>(d_newJobs, d_finishedJobs);

//Launch a host thread to manage incoming jobs
  pthread_t IncomingJobManager = start_IncomingJobsManager(d_newJobs);

//Launch a host thread to manage results from jobs
  pthread_t ResultsManager = start_ResultsManager(d_finishedJobs);

  hipDeviceSynchronize();
  printf("Kernel ended\n");
//wait for the managers to finish (they should never finish)
  void * r;
  pthread_join(IncomingJobManager, &r);
  pthread_join(ResultsManager, &r);

  printf("Both managers have finished\n Main is exiting\n");

  hipStreamDestroy(stream_kernel);
  hipStreamDestroy(stream_dataIn);
  hipStreamDestroy(stream_dataOut);

  DisposeQueue(d_newJobs);
  DisposeQueue(d_finishedJobs);

  return 0;    
}
