#include <stdio.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

hipStream_t stream_dataIn, stream_dataOut; //this is probably not the best way to make these
                                            // streams globally known, oh well.
int SLEEP_TIME;
int NUMBER_OF_JOBS;

#include "Queues/QueueJobs.cu"
#include "IncomingJobsManager.cu"
#include "ResultsManager.cu"
#include "SuperKernel.cu"

////////////////////////////////////////////////////////////////////
// The Main
////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{
//Define constants
  int warp_size = 32;

  int warps = 32;   //possible input arguements
  int blocks = 7;
  NUMBER_OF_JOBS = 224;
  SLEEP_TIME = 1000;
  if(argc>4){
    warps = atoi(argv[1]);
    blocks = atoi(argv[2]);
    NUMBER_OF_JOBS = atoi(argv[3]);
    SLEEP_TIME = atoi(argv[4]);
  }
  
  dim3 threads(warp_size*warps, 1);
  dim3 grid(blocks, 1);

//Allocate streams, Queues
  /* Notes on these structures:
       Streams: one stream for each direction of data movement and one
                    for the Super Kernel.
       QueueJobs: The Scheduler will Enqueue jobDescriptions
                  The Super Kernel will Dequeue and execute them
       QueueResults: The Super Kernel will Enqueue jobResults
                     The Scheduler will Dequeue them and send results to its caller
  */
  hipStream_t stream_kernel;
  hipStreamCreate(&stream_kernel);
  hipStreamCreate(&stream_dataIn);
  hipStreamCreate(&stream_dataOut);

  Queue d_newJobs = CreateQueue(25600); //FIX, make this use stream_dataIn

  Queue d_finishedJobs = CreateQueue(25600); //FIX, make this use stream_dataOut


//Launch the super kernel
  superKernel<<< grid, threads, 0, stream_kernel>>>(d_newJobs, d_finishedJobs);

//Launch a host thread to manage incoming jobs
  pthread_t IncomingJobManager = start_IncomingJobsManager(d_newJobs);

//Launch a host thread to manage results from jobs
  pthread_t ResultsManager = start_ResultsManager(d_finishedJobs);


//wait for the managers to finish (they should never finish)
  void * r;
  pthread_join(IncomingJobManager, &r);
  pthread_join(ResultsManager, &r);

  printf("Both managers have finished\n");
  printf("Destroying Streams...\n");
  hipStreamDestroy(stream_kernel);
  hipStreamDestroy(stream_dataIn);
  hipStreamDestroy(stream_dataOut);

  printf("Destorying Queues...\n");

  DisposeQueue(d_newJobs);

  DisposeQueue(d_finishedJobs);

  printf("Exiting Main\n\n");

  return 0;    
}
